#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAGuard.h>
#include "definitions.h"
#include "diagonal_utils.h"
#include "diagonal_sum.h"

template<typename T>
__global__
void
sum_diagonal_kernel(const T* input, T* output, int64_t flatten_dim, int64_t len) {
    extern __shared__ char shared_bytes[];
    T* sdata = reinterpret_cast<T*>(shared_bytes);

    int64_t tid = threadIdx.x;
    int64_t batch_idx = blockIdx.x;
    int64_t i = threadIdx.x;

    if (i < len) {
        sdata[tid] = input[batch_idx * len * len + i * len + i];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();

    #pragma unroll
    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[batch_idx] = sdata[0];
    }
}


torch::Tensor
sum_diagonal(torch::Tensor input) {
    if (!input.is_contiguous()) {
        input = input.contiguous();
    }
    check_tensor(input);

    auto input_shape = input.sizes();
    int64_t len = input.size(input.dim() - 1);
    input = input.view({-1, len, len});
    int64_t flatten_dim = input.size(0);

    auto output_options = torch::TensorOptions().device(input.device()).dtype(input.dtype());
    torch::Tensor output = torch::zeros({flatten_dim}, output_options);

    c10::cuda::CUDAStream stream = c10::cuda::getCurrentCUDAStream(input.device().index());

    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::BFloat16, input.scalar_type(), "sum_diagonal", [&]{
        auto* input_ptr = input.data_ptr<scalar_t>();
        auto* output_ptr = output.data_ptr<scalar_t>();

        size_t shared_mem_size = THREAD_COUNT * sizeof(scalar_t);

        sum_diagonal_kernel<scalar_t><<<flatten_dim, THREAD_COUNT, shared_mem_size, stream>>>(input_ptr, output_ptr, flatten_dim, len);
    });

    auto final_shape = input_shape.slice(0, input_shape.size() - 2);
    return output.view(final_shape);
}