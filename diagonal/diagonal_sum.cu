#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAGuard.h>
#include "definitions.h"
#include "diagonal_utils.h"
#include "diagonal_sum.h"


template<typename T>
__global__
void
sum_diagonal_kernel(const T* input, T* output, int64_t flatten_dim, int64_t len) {
    extern __shared__ char shared_bytes[];
    T* sdata = reinterpret_cast<T*>(shared_bytes);

    int64_t tid = threadIdx.x;
    int64_t batch_idx = blockIdx.x;
    int64_t i = threadIdx.x;

    if (i < len) {
        sdata[tid] = input[batch_idx * len * len + i * len + i];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();

    #pragma unroll
    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[batch_idx] = sdata[0];
    }
}


torch::Tensor
sum_diagonal(torch::Tensor input) {
    check_tensor(input);
    input = input.contiguous();
    auto input_shape = input.sizes();
    int64_t len = input.size(input.dim() - 1);
    input = input.view({-1, len, len});
    int64_t flatten_dim = input.size(0);
    torch::Tensor output = torch::empty({flatten_dim}, input.options());

    c10::cuda::CUDAStream stream = c10::cuda::getCurrentCUDAStream(input.device().index());

    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::BFloat16, input.scalar_type(), "sum_diagonal", [&]{
        auto* input_ptr = input.data_ptr<scalar_t>();
        auto* output_ptr = output.data_ptr<scalar_t>();

        // Use more threads for better performance on large matrices
        size_t shared_mem_size = (THREAD_COUNT / 32) * sizeof(scalar_t); // One element per warp

        sum_diagonal_kernel<scalar_t><<<flatten_dim, THREAD_COUNT, shared_mem_size, stream>>>(input_ptr, output_ptr, flatten_dim, len);
    });

    auto final_shape = input_shape.slice(0, input_shape.size() - 2);
    hipStreamSynchronize(stream);
    return output.view(final_shape);
}